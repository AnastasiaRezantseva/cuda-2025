#include "hip/hip_runtime.h"
#include "gelu_cuda.h"
#include <chrono>
#include <iostream>
#include <hip/hip_runtime.h>

__global__ void gelu_kernel(const float* input, float* output, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        float x = input[idx];
        float x3 = x * x * x;
        float arg = x + 0.044715f * x3;
        float tanh_arg = 0.797884f * arg;
        float gelu_cdf = 0.5f * (1.0f + tanhf(tanh_arg));
        output[idx] = x * gelu_cdf;
    }
}


std::vector<float> GeluCUDA(const std::vector<float>& input) {
    const int size = input.size();
    std::vector<float> output(size);
    float* d_input;
    float* d_output;

    hipMalloc((void**)&d_input, size * sizeof(float));
    hipMalloc((void**)&d_output, size * sizeof(float));

    hipMemcpy(d_input, input.data(), size * sizeof(float), hipMemcpyHostToDevice);

    int block_size = 256;
    int num_blocks = (size + block_size - 1) / block_size;
    gelu_kernel<<<num_blocks, block_size>>>(d_input, d_output, size);

    hipMemcpy(output.data(), d_output, size * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);

    return output;
}
