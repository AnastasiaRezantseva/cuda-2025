#include "hip/hip_runtime.h"
﻿#include "gelu_cuda.h"
#include <hip/hip_runtime.h>

#define PI 0.797884f
#define BLOCK_SIZE 256

__global__ void GeluKernel(const float* in, float* out, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        float x = in[idx];
        float x_cubed = x * x * x;
        float tanh_input = PI * (x + 0.044715f * x_cubed);
        out[idx] = 0.5f * x * (1.f + tanhf(tanh_input));
    }
}

__host__ std::vector<float> GeluCUDA(const std::vector<float>& input) {
    int size = input.size(), memory = size * sizeof(float);
    int half_size = size >> 1;
    float* in, *out;
	std::vector<float> result(size);

    hipStream_t stream1, stream2;
    hipStreamCreate(&stream1);
    hipStreamCreate(&stream2);
    
    hipMalloc(&in, memory);
    hipMalloc(&out, memory);

    hipMemcpyAsync(in, input.data(), size * 2, hipMemcpyHostToDevice, stream1);
    hipMemcpyAsync(in, input.data(), size * 2, hipMemcpyHostToDevice, stream2);

    GeluKernel<<<(size + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE, 0, stream1>>> (in, out, half_size);
    GeluKernel<<<(size + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE, 0, stream2>>> (in + half_size, out + half_size, half_size);
    
    hipMemcpyAsync(result.data(), out, half_size * sizeof(float), hipMemcpyDeviceToHost, stream1);
    hipMemcpyAsync(result.data() + half_size, out + half_size, half_size * sizeof(float), hipMemcpyDeviceToHost, stream2);

    hipStreamSynchronize(stream1);
    hipStreamSynchronize(stream2);
    hipStreamDestroy(stream1);
    hipStreamDestroy(stream2);
    hipFree(in);
    hipFree(out);

    return result;
}
