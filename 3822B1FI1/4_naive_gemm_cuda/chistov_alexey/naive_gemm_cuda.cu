#include "hip/hip_runtime.h"
#include "naive_gemm_cuda.h"
#include <hip/hip_runtime.h>

__global__ void MatrixMultiply(float* A, float* B, float* C, int n) {
    int rowIndex = blockIdx.y * blockDim.y + threadIdx.y;
    int colIndex = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (rowIndex < n && colIndex < n) {
        float resultValue = 0.0f;
        for (int k = 0; k < n; ++k) {
            resultValue += A[rowIndex * n + k] * B[k * n + colIndex];
        }
        C[rowIndex * n + colIndex] = resultValue;
    } 
}

std::vector<float> NaiveGemmCUDA(const std::vector<float>& a,
                                  const std::vector<float>& b,
                                  int n) {
    std::vector<float> result(n * n);
    float *A, *B, *C;
    int size = n * n * sizeof(float);

    hipMalloc(&A, size);
    hipMalloc(&B, size);
    hipMalloc(&C, size);

    hipMemcpy(A, a.data(), size, hipMemcpyHostToDevice);
    hipMemcpy(B, b.data(), size, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(32, 32);
    dim3 numBlocks((n + 31) / 32, (n + 31) / 32);

    MatrixMultiply<<<numBlocks, threadsPerBlock>>>(A, B, C, n);\
    hipMemcpy(result.data(), C, size, hipMemcpyDeviceToHost);
    
    hipFree(A);
    hipFree(B);
    hipFree(C);
    
    return result;
}