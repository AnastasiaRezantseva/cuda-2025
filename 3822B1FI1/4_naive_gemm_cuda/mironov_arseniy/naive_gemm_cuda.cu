#include "hip/hip_runtime.h"
#include "naive_gemm_cuda.h"
#include <hip/hip_runtime.h>
#define CUDA_BLOCK_SIZE 32

__global__ void SmallMultKernel(const float* in1, const float* in2, float* out, int n) {

    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < n && col < n) {
        float value = 0.0f;
        for (int k = 0; k < n; ++k) {
            value += in1[row * n + k] * in2[k * n + col];
        }
        out[row * n + col] = value;
    }
}


__global__ void MultKernel(const float* in1, const float* in2, float* out, int n) {
    
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row >= n || col >= n) return;
    
    // shared memory
    __shared__ float s_in1[CUDA_BLOCK_SIZE][CUDA_BLOCK_SIZE];
    __shared__ float s_in2[CUDA_BLOCK_SIZE][CUDA_BLOCK_SIZE];

    float res = 0.0f;

    for (int k = 0; k < (n + CUDA_BLOCK_SIZE - 1) / CUDA_BLOCK_SIZE; ++k) {

        s_in1[threadIdx.y][threadIdx.x] = in1[row * n + k * CUDA_BLOCK_SIZE + threadIdx.x];
        s_in2[threadIdx.y][threadIdx.x] = in2[(k * CUDA_BLOCK_SIZE + threadIdx.y) * n + col];
        
        __syncthreads();
        for (int t = 0; t < CUDA_BLOCK_SIZE; ++t) {
            res += s_in1[threadIdx.y][t] * s_in2[t][threadIdx.x];
        }
        __syncthreads();  
    }

    if (row < n && col < n) {
        out[row * n + col] = res;
    }
}

__host__ std::vector<float> NaiveGemmCUDA(const std::vector<float>& a,
                                 const std::vector<float>& b,
                                 int n) {
    int memory = n * n * sizeof(float);
    float* in1, *in2, *out;
	std::vector<float> result(n * n);

    hipMalloc(&in1, memory);
    hipMalloc(&in2, memory);
    hipMalloc(&out, memory);

    hipMemcpy(in1, a.data(), memory, hipMemcpyKind::hipMemcpyHostToDevice);
    hipMemcpy(in2, b.data(), memory, hipMemcpyKind::hipMemcpyHostToDevice);
    
    int grid = (n + CUDA_BLOCK_SIZE - 1) / CUDA_BLOCK_SIZE;
    dim3 gridSize(grid, grid),
         blockSize(CUDA_BLOCK_SIZE, CUDA_BLOCK_SIZE);
    if(n < CUDA_BLOCK_SIZE){
        SmallMultKernel<<<gridSize, blockSize>>> (in1, in2, out, n);
    }
    else{
        MultKernel<<<gridSize, blockSize>>> (in1, in2, out, n);
    }
    hipMemcpy(result.data(), out, memory, hipMemcpyKind::hipMemcpyDeviceToHost);

    hipFree(in1);
    hipFree(in2);
    hipFree(out);

    return result;
}
