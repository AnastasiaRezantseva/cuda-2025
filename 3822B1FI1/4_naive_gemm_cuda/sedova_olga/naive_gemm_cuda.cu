#include "hip/hip_runtime.h"
#include "naive_gemm_cuda.h"
#include <hip/hip_runtime.h>
#include <stdexcept>

__global__ void naive_gemm_kernel(const float* a, const float* b, float* c, int n) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < n && col < n) {
        float sum = 0.0f;
        for (int k = 0; k < n; ++k) {
            sum += a[row * n + k] * b[k * n + col];
        }
        c[row * n + col] = sum;
    }
}

std::vector<float> NaiveGemmCUDA(const std::vector<float>& a,
                                 const std::vector<float>& b,
                                 int n) {
    if (n == 0) return {};

    float *d_a = nullptr, *d_b = nullptr, *d_c = nullptr;
    size_t size = n * n * sizeof(float);

    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_c, size);

    hipMemcpy(d_a, a.data(), size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b.data(), size, hipMemcpyHostToDevice);

    dim3 blockSize(16, 16);
    dim3 gridSize((n + blockSize.x - 1) / blockSize.x,
                  (n + blockSize.y - 1) / blockSize.y);

    naive_gemm_kernel<<<gridSize, blockSize>>>(d_a, d_b, d_c, n);
    hipDeviceSynchronize();

    std::vector<float> c(n * n);
    hipMemcpy(c.data(), d_c, size, hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return c;
}
