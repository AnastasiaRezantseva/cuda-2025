#include "hip/hip_runtime.h"
#include "block_gemm_cuda.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <cstdlib>

const int block_size = 16;

__global__ void kernel(const float* a, const float* b, float* const c, const int size) {
	__shared__ float sa[16][16];
	__shared__ float sb[16][16];
	int row = blockIdx.y * block_size + threadIdx.y;
	int col = blockIdx.x * block_size + threadIdx.x;
	float elem = 0.0f;
	for (int k = 0; k < n; k+= block_size) {
		if (row < n && k + threadIdx.y < n) {
			sa[threadIdx.y][threadIdx.x] = a[row * n + k + threadIdx.x];
		}
		else {
			sa[threadIdx.y][threadIdx.x] = 0.0f;
		}
		if (col < n && k + threadIdx.x < n) {
			sb[threadIdx.y][threadIdx.x] = b[col + n * (k + threadIdx.y)];
		}
		else {
			sb[threadIdx.y][threadIdx.x] = 0.0f;
		}
		__syncthreads();
		for (int i = 0; i < block_size; ++i) {
			elem += sa[threadIdx.y][i] * sb[i][threadIdx.x];
		}
		__syncthreads();
	}
	if (row < n && col < n) {
        c[row * n + col] = elem;
    }
}


std::vector<float> BlockGemmCUDA(const std::vector<float>& a,
                                 const std::vector<float>& b,
                                 int n) {
    std::vector<float> c(n * n, 0.0f);
	size_t bytes = n * n * sizeof(float);
	float* device_a;
	float* device_b;
	float* device_c;
	hipMalloc(&device_a, bytes);
	hipMalloc(&device_b, bytes);
	hipMalloc(&device_c, bytes);
	hipMemcpy(device_a, a.data(), bytes, hipMemcpyHostToDevice);
	hipMemcpy(device_b, b.data(), bytes, hipMemcpyHostToDevice);
	dim3 dimBlock(block_size, block_size);
	dim3 dimGrid((n + block_size - 1) / block_size, (n + block_size - 1) / block_size);
	kernel<<<dimGrid, dimBlock>>>(device_a, device_b, device_c, n, block_size);
	hipMemcpy(c.data(), device_c, bytes, hipMemcpyDeviceToHost);
	hipFree(device_a);
	hipFree(device_b);
	hipFree(device_c);
	return c;
}