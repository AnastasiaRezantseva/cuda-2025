#include "hip/hip_runtime.h"
#include "block_gemm_cuda.h"
#include <hip/hip_runtime.h>
#define CUDA_BLOCK_SIZE 16

__global__ void MultKernel(const float* in1, const float* in2, float* out, int n) {
    
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if (row >= n || col >= n) return;
    
    // shared memory
    __shared__ float s_in1[CUDA_BLOCK_SIZE][CUDA_BLOCK_SIZE];
    __shared__ float s_in2[CUDA_BLOCK_SIZE][CUDA_BLOCK_SIZE];

    float res = 0.0f;

    for (int k = 0; k < (n + CUDA_BLOCK_SIZE - 1) / CUDA_BLOCK_SIZE; ++k) {

        s_in1[threadIdx.y][threadIdx.x] = in1[row * n + k * CUDA_BLOCK_SIZE + threadIdx.x];
        s_in2[threadIdx.y][threadIdx.x] = in2[(k * CUDA_BLOCK_SIZE + threadIdx.y) * n + col];
        
        __syncthreads();
        for (int t = 0; t < CUDA_BLOCK_SIZE; ++t) {
            res += s_in1[threadIdx.y][t] * s_in2[t][threadIdx.x];
        }
        __syncthreads();  
    }

    if (row < n && col < n) {
        out[row * n + col] = res;
    }
}

__host__ std::vector<float> BlockGemmCUDA(const std::vector<float>& a,
                                 const std::vector<float>& b,
                                 int n) {
    int memory = n * n * sizeof(float);
    float* in1, *in2, *out;
	std::vector<float> result(n * n);
    
    hipMalloc(&in1, memory);
    hipMalloc(&in2, memory);
    hipMalloc(&out, memory);

    hipMemcpy(in1, a.data(), memory, hipMemcpyKind::hipMemcpyHostToDevice);
    hipMemcpy(in2, b.data(), memory, hipMemcpyKind::hipMemcpyHostToDevice);
    
    int grid = (n + CUDA_BLOCK_SIZE - 1) / CUDA_BLOCK_SIZE;
    dim3 gridSize(grid, grid),
         blockSize(CUDA_BLOCK_SIZE, CUDA_BLOCK_SIZE);
    
    MultKernel<<<gridSize, blockSize>>> (in1, in2, out, n);
    hipMemcpy(result.data(), out, memory, hipMemcpyKind::hipMemcpyDeviceToHost);

    hipFree(in1);
    hipFree(in2);
    hipFree(out);

    return result;
}