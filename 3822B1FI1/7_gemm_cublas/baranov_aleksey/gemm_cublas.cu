#include "gemm_cublas.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>

std::vector<float> GemmCUBLAS(const std::vector<float>& matrix_a,
                              const std::vector<float>& matrix_b,
                              int matrix_size) {
    std::vector<float> result(matrix_size * matrix_size);
    size_t matrix_bytes = matrix_size * matrix_size * sizeof(float);

    float* device_a;
    float* device_b;
    float* device_c;
    float* device_c_transposed;

    hipMalloc(&device_a, matrix_bytes);
    hipMalloc(&device_b, matrix_bytes);
    hipMalloc(&device_c, matrix_bytes);
    hipMalloc(&device_c_transposed, matrix_bytes);

    hipblasSetMatrix(matrix_size, matrix_size, sizeof(float), matrix_a.data(), matrix_size, device_a, matrix_size);
    hipblasSetMatrix(matrix_size, matrix_size, sizeof(float), matrix_b.data(), matrix_size, device_b, matrix_size);

    hipblasHandle_t cublas_handle;
    hipblasCreate(&cublas_handle);

    const float alpha = 1.0f;
    const float beta = 0.0f;

    hipblasSgemm(cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_T, matrix_size, matrix_size, matrix_size,
                &alpha, device_a, matrix_size, device_b, matrix_size, &beta, device_c, matrix_size);

    hipblasSgeam(cublas_handle, HIPBLAS_OP_T, HIPBLAS_OP_N, matrix_size, matrix_size,
                &alpha, device_c, matrix_size, &beta, nullptr, matrix_size, device_c_transposed, matrix_size);

    hipblasGetMatrix(matrix_size, matrix_size, sizeof(float), device_c_transposed, matrix_size, result.data(), matrix_size);

    hipFree(device_b);
    hipFree(device_c);
    hipFree(device_a);
    hipFree(device_c_transposed);

    hipblasDestroy(cublas_handle);

    return result;
}