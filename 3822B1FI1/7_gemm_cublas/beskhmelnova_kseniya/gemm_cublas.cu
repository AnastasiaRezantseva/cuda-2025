#include "gemm_cublas.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <vector>
#include <iostream>

std::vector<float> GemmCUBLAS(const std::vector<float>& a,
	const std::vector<float>& b,
	int n) {
	size_t size = n * n * sizeof(float);
	std::vector<float> c(n * n, 0.0f);

	float* d_A, * d_B, * d_C;
	hipMalloc((void**)&d_A, size);
	hipMalloc((void**)&d_B, size);
	hipMalloc((void**)&d_C, size);

	hipMemcpy(d_A, a.data(), size, hipMemcpyHostToDevice);
	hipMemcpy(d_B, b.data(), size, hipMemcpyHostToDevice);

	hipblasHandle_t handle;
	hipblasCreate(&handle);

	const float alpha = 1.0f, beta = 0.0f;

	hipblasSgemm(handle,
		HIPBLAS_OP_N, HIPBLAS_OP_N,
		n, n, n,
		&alpha,
		d_A, n,
		d_B, n,
		&beta,
		d_C, n);

	hipMemcpy(c.data(), d_C, size, hipMemcpyDeviceToHost);

	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	hipblasDestroy(handle);

	return c;
}