#include "gemm_cublas.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <vector>

std::vector<float> GemmCUBLAS(const std::vector<float>& a,
                              const std::vector<float>& b,
                              int n) {
    int m_size = n * n * sizeof(float);
    std::vector<float> res(n * n);

    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, m_size);
    hipMalloc(&d_B, m_size);
    hipMalloc(&d_C, m_size);

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    hipblasSetMatrix(n, n, sizeof(float), a.data(), n, d_A, n);
    hipblasSetMatrix(n, n, sizeof(float), b.data(), n, d_B, n);

    const float alpha = 1.0f;
    const float beta = 0.0f;

    hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, n, n, n, &alpha, d_A, n, d_B, n, &beta, d_C, n);

    hipblasGetMatrix(n, n, sizeof(float), d_C, n, res.data(), n);

    hipblasDestroy(handle);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return res;
}
