#include "gemm_cublas.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <vector>

std::vector<float> GemmCUBLAS(const std::vector<float>& a,
                              const std::vector<float>& b,
                              int n) {
    int size = n * n * sizeof(float);
    std::vector<float> c(n * n);

    float *d_A, *d_B, *d_C, *d_C_transposed;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);
    hipMalloc(&d_C_transposed, size);

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    hipblasSetMatrix(n, n, sizeof(float), a.data(), n, d_A, n);
    hipblasSetMatrix(n, n, sizeof(float), b.data(), n, d_B, n);

    const float alpha = 1.0f;
    const float beta = 0.0f;

    hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, n, n, n, &alpha, d_A, n, d_B, n, &beta, d_C, n);
    hipblasSgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, n, n, &alpha, d_C, n, &beta, nullptr, n, d_C_transposed, n);
    hipblasGetMatrix(n, n, sizeof(float), d_C_transposed, n, c.data(), n);

    hipblasDestroy(handle);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipFree(d_C_transposed);

    return c;
}
