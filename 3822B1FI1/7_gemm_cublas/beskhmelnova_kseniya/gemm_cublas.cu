#include "gemm_cublas.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <vector>
#include <iostream>

std::vector<float> GemmCUBLAS(const std::vector<float>& a,
	const std::vector<float>& b,
	int n) {
	int size = n * n * sizeof(float);
	std::vector<float> c(n * n, 0.0f);

	float* d_A, * d_B, * d_C;
	hipMalloc((void**)&d_A, size);
	hipMalloc((void**)&d_B, size);
	hipMalloc((void**)&d_C, size);

	hipMemcpy(d_A, a.data(), size, hipMemcpyHostToDevice);
	hipMemcpy(d_B, b.data(), size, hipMemcpyHostToDevice);

	hipblasHandle_t handle;
	hipblasCreate(&handle);

	const float alpha = 1.0f, beta = 0.0f;

	hipblasSgemm(handle, 
             HIPBLAS_OP_T, HIPBLAS_OP_T,
             n, n, n, 
             &alpha, 
             d_B, n,
             d_A, n, 
             &beta, 
             d_C, n);

	hipMemcpy(c.data(), d_C, size, hipMemcpyDeviceToHost);

	for (int i = 0; i < n; i++) {
		for (int j = i + 1; j < n; j++) {
			std::swap(c[i * n + j], c[j * n + i]);
		}
	}

	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	hipblasDestroy(handle);

	return c;
}