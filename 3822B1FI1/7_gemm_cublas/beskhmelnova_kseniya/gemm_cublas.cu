#include "gemm_cublas.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <vector>

std::vector<float> GemmCUBLAS(const std::vector<float>& a,
                              const std::vector<float>& b,
                              int n) {
    int size = n * n * sizeof(float);
    std::vector<float> c(n * n);

    float *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    hipblasSetMatrix(n, n, sizeof(float), a.data(), n, d_A, n);
    hipblasSetMatrix(n, n, sizeof(float), b.data(), n, d_B, n);

    const float alpha = 1.0f;
    const float beta = 0.0f;

    hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, n, n, n, &alpha, d_A, n, d_B, n, &beta, d_C, n);

    hipblasGetMatrix(n, n, sizeof(float), d_C, n, c.data(), n);

    hipblasDestroy(handle);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return c;
}
