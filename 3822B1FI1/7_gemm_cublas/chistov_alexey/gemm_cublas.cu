#include "gemm_cublas.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>

std::vector<float> GemmCUBLAS(const std::vector<float>& a,
                              const std::vector<float>& b,
                              int n) {
    std::vector<float> result(n * n);
    size_t size = n * n * sizeof(float);
    float* d_a;
    float* d_b;
    float* d_c;
    float *d_c_transposed;

    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_c, size);
    hipMalloc(&d_c_transposed, size);
    hipblasSetMatrix(n, n, sizeof(float), a.data(), n, d_a, n);
    hipblasSetMatrix(n, n, sizeof(float), b.data(), n, d_b, n);

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    const float alpha = 1.0f;
    const float beta = 0.0f;

    hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, n, n, n, &alpha, d_a, n, d_b, n, &beta, d_c, n);
    hipblasSgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, n, n, &alpha, d_c, n, &beta, nullptr, n, d_c_transposed, n);
    hipblasGetMatrix(n, n, sizeof(float), d_c_transposed, n, result.data(), n);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipFree(d_c_transposed);
    hipblasDestroy(handle);

    return result;
}