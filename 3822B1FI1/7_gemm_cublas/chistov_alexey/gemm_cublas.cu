#include "gemm_cublas.h"
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <iostream>

std::vector<float> GemmCUBLAS(const std::vector<float>& a,
                              const std::vector<float>& b, int size) {
    std::vector<float> result(size * size);

    size_t size = size * size * sizeof(float);

    float* d_a;
    float* d_b;
    float* d_c;

    if (hipMalloc(&d_a, size) != hipSuccess) {
        std::cerr << "Error allocating device memory for d_a" << std::endl;
        return result;
    }
    if (hipMalloc(&d_b, size) != hipSuccess) {
        std::cerr << "Error allocating device memory for d_b" << std::endl;
        hipFree(d_a);
        return result;
    }
    if (hipMalloc(&d_c, size) != hipSuccess) {
        std::cerr << "Error allocating device memory for d_c" << std::endl;
        hipFree(d_a);
        hipFree(d_b);
        return result;
    }

    hipMemcpy(d_a, a.data(), size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b.data(), size, hipMemcpyHostToDevice);

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    const float alpha = 1.0f;
    const float beta = 0.0f;
    hipblasSetMathMode(handle, HIPBLAS_TF32_TENSOR_OP_MATH);

    hipblasStatus_t status = hipblasGemmEx(handle,
                 HIPBLAS_OP_N, HIPBLAS_OP_N,
                 size, size, size,
                 &alpha,
                 d_b, HIP_R_32F, size,
                 d_a, HIP_R_32F, size,
                 &beta,
                 d_c, HIP_R_32F, size,
                 HIPBLAS_COMPUTE_32F_FAST_16F, HIPBLAS_GEMM_DEFAULT);

    if (status != HIPBLAS_STATUS_SUCCESS) {
        std::cerr << "Error during cuBLAS GEMM operation" << std::endl;
    }

    hipMemcpy(result.data(), d_c, size, hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipblasDestroy(handle);

    return result;
}