#include "gemm_cublas.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"

std::vector<float> GemmCUBLAS(const std::vector<float>& a,
                              const std::vector<float>& b,
                              int n) {
  std::vector<float> c(n * n, 0.0f);
  size_t bytes = n * n * sizeof(float);
  float* device_a;
  float* device_b;
  float* device_c;
  hipMalloc(&device_a, bytes);
  hipMalloc(&device_b, bytes);
  hipMalloc(&device_c, bytes);
  hipMemcpy(device_a, a.data(), bytes, hipMemcpyHostToDevice);
  hipMemcpy(device_b, b.data(), bytes, hipMemcpyHostToDevice);

  const float alpha = 1.0f;
  const float beta = 0.0f;
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, n, n, n, &alpha, device_b, n, device_a, n, &beta, device_c, n);
  hipblasDestroy(handle);

  hipMemcpy(c.data(), device_c, bytes, hipMemcpyDeviceToHost);
  hipFree(device_a);
  hipFree(device_b);
  hipFree(device_c);

  return c;
}