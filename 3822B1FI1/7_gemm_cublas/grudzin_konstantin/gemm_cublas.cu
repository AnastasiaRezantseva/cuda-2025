#include "gemm_cublas.h"
#include <hipblas.h>
#include <hip/hip_runtime.h>

std::vector<float> GemmCUBLAS(const std::vector<float>& a,
                              const std::vector<float>& b,
                              int n) {
    std::vector<float> C(n*n);
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    float *d_A, *d_B, *d_C, *d_C_T;
    hipMalloc(&d_A, n * n * sizeof(float));
    hipMalloc(&d_B, n * n * sizeof(float));
    hipMalloc(&d_C, n * n * sizeof(float));
    hipMalloc(&d_C_T, n * n * sizeof(float));

    hipMemcpy(d_A, a.data(), n * n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_B, b.data(), n * n * sizeof(float), hipMemcpyHostToDevice);

    float alpha = 1.0f, beta = 0.0f;
    hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, n, n, n, &alpha, d_A, n, d_B, n, &beta, d_C, n);
    hipblasSgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, n, n, &alpha, d_C, n, &beta, nullptr, n, d_C_T, n);

    hipMemcpy(C.data(), d_C_T, n * n * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipFree(d_C_T);
    hipblasDestroy(handle);
    return C;
}