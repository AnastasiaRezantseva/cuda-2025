#include <hip/hip_runtime.h>
#include <hipblas.h>
#include "gemm_cublas.h"

std::vector<float> GemmCUBLAS(const std::vector<float>& a,
  const std::vector<float>& b,
  int n) {
  std::vector<float> result(n * n, 0.f);
  int size = n * n * sizeof(float);
  float* A, *B, *C,* Ct;  // Ct - transposed matrix C

  hipMalloc(&A, size);
  hipMalloc(&B, size);
  hipMalloc(&C, size);
  hipMalloc(&Ct, size);

  hipMemcpy(A, a.data(), size, hipMemcpyHostToDevice);
  hipMemcpy(B, b.data(), size, hipMemcpyHostToDevice);

  hipblasHandle_t handle;
  hipblasCreate(&handle);

  const float alpha = 1.f;
  const float beta = 0.f;

  hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, n, n, n, &alpha, B, n, A, n, &beta, C, n);
  hipblasSgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, n, n, &alpha, C, n, &beta, nullptr, n, Ct, n);

  hipblasDestroy(handle);

  hipMemcpy(result.data(), Ct, size, hipMemcpyDeviceToHost);

  hipFree(A);
  hipFree(B);
  hipFree(C);
  hipFree(Ct);

  return result;
}