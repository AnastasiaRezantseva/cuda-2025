#include "gemm_cublas.h"

std::vector<float> GemmCUBLAS(const std::vector<float>& a, const std::vector<float>& b, int n) {
  int m_size = n * n * sizeof(float);
  std::vector<float> res(n * n);

  float *a_m, *b_m, *res_m, *res_m_t;
  hipMalloc(&a_m, m_size);
  hipMalloc(&b_m, m_size);
  hipMalloc(&res_m, m_size);
  hipMalloc(&res_m_t, m_size);

  hipblasHandle_t handle;
  hipblasCreate(&handle);

  hipblasSetMatrix(n, n, sizeof(float), a.data(), n, a_m, n);
  hipblasSetMatrix(n, n, sizeof(float), b.data(), n, b_m, n);

  float alpha = 1.0f;
  float beta = 0.0f;

  hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, n, n, n, &alpha, a_m, n, b_m, n, &beta, res_m, n);
  hipblasSgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, n, n, &alpha, res_m, n, &beta, nullptr, n, res_m_t, n);

  hipblasGetMatrix(n, n, sizeof(float), res_m_t, n, res.data(), n);

  hipblasDestroy(handle);

  hipFree(a_m);
  hipFree(b_m);
  hipFree(res_m);
  hipFree(res_m_t);

  return res;
}