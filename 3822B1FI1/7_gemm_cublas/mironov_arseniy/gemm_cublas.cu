#include "gemm_cublas.h"
#include <hipblas.h>
#include <hip/hip_runtime.h>


std::vector<float> GemmCUBLAS(const std::vector<float>& a,
                              const std::vector<float>& b,
                              int n) {
    int memory = n * n * sizeof(float);
    int square = n * n;
    float* in1, *in2, *out, *out_transposed;
	std::vector<float> result(square);
    
    hipMalloc(&in1, memory);
    hipMalloc(&in2, memory);
    hipMalloc(&out, memory);
    hipMalloc(&out_transposed, memory);

    hipMemcpy(in1, a.data(), memory, hipMemcpyHostToDevice);
    hipMemcpy(in2, b.data(), memory, hipMemcpyHostToDevice);

    hipblasHandle_t handle;
    hipblasCreate(&handle);
    float alpha = 1.0f, beta= 0.0f;

    hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, n, n, n, &alpha, in1, n, in2, n, &beta, out, n);
    hipblasSgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, n, n, &alpha, out, n, &beta, nullptr, n, out_transposed, n);
    hipMemcpy(result.data(), out_transposed, memory, hipMemcpyDeviceToHost);
    
    hipFree(in1);
    hipFree(in2);
    hipFree(out);
    hipFree(out_transposed);
    hipblasDestroy(handle);
    return result;
}