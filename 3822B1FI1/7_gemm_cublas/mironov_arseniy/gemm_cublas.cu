#include "gemm_cublas.h"
#include <hipblas.h>
#include <hip/hip_runtime.h>

std::vector<float> GemmCUBLAS(const std::vector<float>& a,
                              const std::vector<float>& b,
                              int n) {
    int memory = n * n * sizeof(float);
    int square = n * n;
    float* in1, *in2, *out;
	std::vector<float> result(square);
    
    hipMalloc(&in1, memory);
    hipMalloc(&in2, memory);
    hipMalloc(&out, memory);

    hipMemcpy(in1, a.data(), memory, hipMemcpyHostToDevice);
    hipMemcpy(in2, b.data(), memory, hipMemcpyHostToDevice);

    hipblasHandle_t handle;
    hipblasCreate(&handle);
    float alpha = 1.0f, beta= 0.0f;

    hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, n, n, n, &alpha, in1, n, in2, n, &beta, out, n);

    hipMemcpy(result.data(), out, memory, hipMemcpyDeviceToHost);
    
    for(int i = 0; i < n; ++i){
        for(int j = i + 1; j < n; ++j){
            std::swap(result[i * n + j], result[j * n + i]);
        }
    }
    hipFree(in1);
    hipFree(in2);
    hipFree(out);
    hipblasDestroy(handle);
    return result;
}
