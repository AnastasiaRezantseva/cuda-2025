#include "gemm_cublas.h"
#include <cstring>
#include <hipblas.h>
#include <hip/hip_runtime.h>

std::vector<float> GemmCUBLAS(const std::vector<float>& a,
                              const std::vector<float>& b,
                              int n) {
    size_t required_size = static_cast<size_t>(n) * n;

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    hipStream_t stream;
    hipStreamCreate(&stream);
    hipblasSetStream(handle, stream);

    size_t memory = required_size * sizeof(float);
    float* h_A_pinned, * h_B_pinned, * h_C_pinned;
    hipHostAlloc(&h_A_pinned, memory, hipHostMallocDefault);
    hipHostAlloc(&h_B_pinned, memory, hipHostMallocDefault);
    hipHostAlloc(&h_C_pinned, memory, hipHostMallocDefault);

    std::memcpy(h_A_pinned, a.data(), memory);
    std::memcpy(h_B_pinned, b.data(), memory);

    float* d_A, * d_B, * d_C;
    hipMalloc(&d_A, memory);
    hipMalloc(&d_B, memory);
    hipMalloc(&d_C, memory);
    hipMemcpyAsync(d_A, h_A_pinned, memory, hipMemcpyHostToDevice, stream);
    hipMemcpyAsync(d_B, h_B_pinned, memory, hipMemcpyHostToDevice, stream);

    const float alpha = 1.0f;
    const float beta = 0.0f;
    hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, n, n, n, &alpha, d_B, n, d_A, n, &beta, d_C, n);
    hipMemcpyAsync(h_C_pinned, d_C, memory, hipMemcpyDeviceToHost, stream);
    hipStreamSynchronize(stream);

    std::vector<float> result(n * n);
    std::memcpy(result.data(), h_C_pinned, memory);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    hipHostFree(h_A_pinned);
    hipHostFree(h_B_pinned);
    hipHostFree(h_C_pinned);
    hipStreamDestroy(stream);
    hipblasDestroy(handle);

    return result;
}