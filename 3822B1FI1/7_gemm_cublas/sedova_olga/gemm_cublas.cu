#include "gemm_cublas.h"
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <stdexcept>
#include <vector>

std::vector<float> GemmCUBLAS(const std::vector<float>& a,
                              const std::vector<float>& b,
                              int n) {
    if (n == 0) return {};

    hipblasHandle_t handle;
    hipblasCreate(&handle);

    size_t size = n * n * sizeof(float);
    float *d_a = nullptr, *d_b = nullptr, *d_c = nullptr;

    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_c, size);

    hipMemcpy(d_a, a.data(), size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b.data(), size, hipMemcpyHostToDevice);

    const float alpha = 1.0f;
    const float beta = 0.0f;

    hipblasStatus_t stat = hipblasSgemm(handle,
                                     HIPBLAS_OP_T, HIPBLAS_OP_T,
                                     n, n, n,
                                     &alpha,
                                     d_b, n,
                                     d_a, n,
                                     &beta,
                                     d_c, n);

    if (stat != HIPBLAS_STATUS_SUCCESS) {
        hipFree(d_a);
        hipFree(d_b);
        hipFree(d_c);
        hipblasDestroy(handle);
        throw std::runtime_error("cuBLAS sgemm failed");
    }

    std::vector<float> c(n * n);
    hipMemcpy(c.data(), d_c, size, hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipblasDestroy(handle);

    return c;
}
