#include "gemm_cublas.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>

std::vector<float> GemmCUBLAS(const std::vector<float>& input_matrix_a,
                              const std::vector<float>& input_matrix_b,
                              int size) {
    size_t total_bytes = size * size * sizeof(float);
    std::vector<float> output_matrix(size * size);

    float* device_matrix_a;
    float* device_matrix_b;
    float* device_matrix_c;

    hipMalloc(&device_matrix_a, total_bytes);
    hipMalloc(&device_matrix_b, total_bytes);
    hipMalloc(&device_matrix_c, total_bytes);

    hipMemcpy(device_matrix_a, input_matrix_a.data(), total_bytes, hipMemcpyHostToDevice);
    hipMemcpy(device_matrix_b, input_matrix_b.data(), total_bytes, hipMemcpyHostToDevice);
    
    hipblasHandle_t cublas_context;
    hipblasCreate(&cublas_context);

    const float scalar_alpha = 1.0f;
    const float scalar_beta = 0.0f;
    
    hipblasSgemm(cublas_context, HIPBLAS_OP_T, HIPBLAS_OP_T, size, size, size,
                &scalar_alpha, device_matrix_a, size, device_matrix_b, size, &scalar_beta, device_matrix_c, size);

    hipMemcpy(output_matrix.data(), device_matrix_c, total_bytes, hipMemcpyDeviceToHost);

    hipFree(device_matrix_a);
    hipFree(device_matrix_b);
    hipFree(device_matrix_c);

    hipblasDestroy(cublas_context);

    return output_matrix;
}
