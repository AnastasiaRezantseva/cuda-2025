#include "hip/hip_runtime.h"
#include "fft_cufft.h"
#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>

__global__ void kernel(float* data, int total_size, float normalization_factor) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < total_size) {
        data[idx] *= normalization_factor;
    }
}

std::vector<float> FffCUFFT(const std::vector<float>& input, int batch) {
    int total_size = input.size();
    int transform_size = total_size / (2 * batch);
    int data_bytes = total_size * sizeof(float);

    hipfftComplex* device_data;
    hipMalloc(&device_data, data_bytes);
    hipMemcpy(device_data, input.data(), data_bytes, hipMemcpyHostToDevice);

    hipfftHandle fft_plan;
    hipfftPlan1d(&fft_plan, transform_size, HIPFFT_C2C, batch);

    hipfftExecC2C(fft_plan, device_data, device_data, HIPFFT_FORWARD);
    hipfftExecC2C(fft_plan, device_data, device_data, HIPFFT_BACKWARD);

    int threads_per_block = 256;
    int blocks_per_grid = (total_size + threads_per_block - 1) / threads_per_block;
    float normalization_factor = 1.0f / static_cast<float>(transform_size);
    kernel<<<blocks_per_grid, threads_per_block>>>(reinterpret_cast<float*>(device_data), total_size, normalization_factor);

    std::vector<float> result(total_size);
    hipMemcpy(result.data(), device_data, data_bytes, hipMemcpyDeviceToHost);

    hipfftDestroy(fft_plan);
    hipFree(device_data);

    return result;
}