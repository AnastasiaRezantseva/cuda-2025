#include "hip/hip_runtime.h"
#include "fft_cufft.h"
#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>

__global__ void normalize_kernel(float* data, int size, float norm_factor) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < size) {
		data[i] *= norm_factor;
	}
}

std::vector<float> FffCUFFT(const std::vector<float>& input, int batch) {
	int size = input.size();

	int n = size / (2 * batch);
	int bytes = size * sizeof(float);

	hipfftComplex* d_data;
	hipMalloc(&d_data, bytes);
	hipMemcpy(d_data, input.data(), bytes, hipMemcpyHostToDevice);

	hipfftHandle plan;
	hipfftPlan1d(&plan, n, HIPFFT_C2C, batch);

	hipfftExecC2C(plan, d_data, d_data, HIPFFT_FORWARD);
	hipfftExecC2C(plan, d_data, d_data, HIPFFT_BACKWARD);

	int threadsPerBlock = 256;
	int blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
	float norm_factor = 1.0f / static_cast<float>(n);
	normalize_kernel << <blocksPerGrid, threadsPerBlock >> > (reinterpret_cast<float*>(d_data), size, norm_factor);

	std::vector<float> result(size);
	hipMemcpy(result.data(), d_data, bytes, hipMemcpyDeviceToHost);

	hipfftDestroy(plan);
	hipFree(d_data);

	return result;
}