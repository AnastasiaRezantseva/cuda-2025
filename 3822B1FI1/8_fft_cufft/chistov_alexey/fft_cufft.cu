#include "hip/hip_runtime.h"
#include "fft_cufft.h"
#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>

__global__ void normalize_kernel(float* data, int n, float normalizationFactor) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        data[i] *= normalizationFactor;
    }
}

std::vector<float> FffCUFFT(const std::vector<float>& input, int batch) {
    int totalSize  = input.size();
    int n = (totalSize  / batch) >> 1;

    int byteSize  = sizeof(hipfftComplex) * n * batch;
    hipfftHandle fftPlan;
    hipfftPlan1d(&fftPlan, n, HIPFFT_C2C, batch);
    hipfftComplex* d_input;

    hipMalloc(&d_input, byteSize );
    hipMemcpy(d_input, input.data(), byteSize , hipMemcpyHostToDevice);
    hipfftExecC2C(fftPlan, d_input, d_input, HIPFFT_FORWARD);
    hipfftExecC2C(fftPlan, d_input, d_input, HIPFFT_BACKWARD);
    hipDeviceProp_t deviceProp;

    hipGetDeviceProperties(&deviceProp, 0);
    size_t threadsPerBlock = deviceProp.maxThreadsPerBlock;
    size_t blocksPerGrid = (totalSize  + threadsPerBlock - 1) / threadsPerBlock;
    float norm = 1.0f / static_cast<float>(n);
    normalize_kernel<<<blocksPerGrid, threadsPerBlock>>>(
        reinterpret_cast<float*>(d_input), totalSize , norm);
    
    std::vector<float> result(totalSize);
    hipMemcpy(result.data(), d_input, byteSize , hipMemcpyDeviceToHost);

    hipfftDestroy(fftPlan);
    hipFree(d_input);

    return result;
}