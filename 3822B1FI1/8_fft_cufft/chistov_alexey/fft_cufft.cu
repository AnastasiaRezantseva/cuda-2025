#include "hip/hip_runtime.h"
#include "fft_cufft.h"
#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>
constexpr int threadsPerBlock = 512;

__global__ void normalize_kernel(float* data, int n, float normalizationFactor) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        data[i] *= normalizationFactor;
    }
}

std::vector<float> FffCUFFT(const std::vector<float>& input, int batch) {
    int totalSize = input.size();
    int fftSize = totalSize / (2 * batch);
    int byteSize = totalSize * sizeof(hipfftComplex);

    hipfftComplex* d_input;
    hipMalloc(&d_input, byteSize);
    hipMemcpy(d_input, input.data(), byteSize, hipMemcpyHostToDevice);

    hipfftHandle fftPlan;
    hipfftPlan1d(&fftPlan, fftSize, HIPFFT_C2C, batch);

    hipfftExecC2C(fftPlan, d_input, d_input, HIPFFT_FORWARD);
    hipfftExecC2C(fftPlan, d_input, d_input, HIPFFT_BACKWARD);

    int blocksPerGrid = (totalSize + threadsPerBlock - 1) / threadsPerBlock;
    float normalizationFactor = 1.0f / static_cast<float>(fftSize);
    normalize_kernel<<<blocksPerGrid, threadsPerBlock>>>(reinterpret_cast<float*>(d_input), totalSize, normalizationFactor);

    std::vector<float> result(totalSize);
    hipMemcpy(result.data(), d_input, byteSize, hipMemcpyDeviceToHost);

    hipfftDestroy(fftPlan);
    hipFree(d_input);

    return result;
}
