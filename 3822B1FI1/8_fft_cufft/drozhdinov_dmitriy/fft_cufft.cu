#include "hip/hip_runtime.h"
#include "fft_cufft.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <>
#include <cstdlib>
#include <vector>

__global__ void kernel(float* input, int size, float norma) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < size) {
	  input[i] *= norma;
  }
}

std::vector<float> FffCUFFT(const std::vector<float>& input, int batch) {
	const int size = input.size();
	std::vector<float> result(size);
	int n = (size / batch) >> 1;
	int bytes = sizeof(hipfftComplex) * n * batch;
	hipfftComplex* data;
	hipMalloc(&data, bytes);
	hipMemcpy(data, input.data(), bytes, hipMemcpyHostToDevice);
	hipfftHandle handle;
	hipfftPlan1d(&handle, n, HIPFFT_C2C, batch);
	hipfftExecC2C(handle, data, data, HIPFFT_FORWARD);
	hipfftExecC2C(handle, data, data, HIPFFT_BACKWARD);
	hipDeviceProp_t deviceProp;
	hipGetDeviceProperties(&deviceProp, 0);
	size_t threadsPerBlock = deviceProp.maxThreadsPerBlock;
	size_t blocksPerGrid = (size + threadsPerBlock - 1) / threadsPerBlock;
	float norma = 1.0f / static_cast<float>(n);
	kernel<<<blocksPerGrid, threadsPerBlock>>>(reinterpret_cast<float*>(data), size, norma);
	hipMemcpy(result.data(), data, bytes, hipMemcpyDeviceToHost);
	hipfftDestroy(handle);
	hipFree(data);
	return result;
}