#include "hip/hip_runtime.h"
#include "fft_cufft.h"
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <iostream>
#define BLOCK_SIZE 256

__global__ void normalizeKernel(hipfftComplex* data, int totalComplex, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < totalComplex) {
        data[idx].x /= n;
        data[idx].y /= n;
    }
}

std::vector<float> FffCUFFT(const std::vector<float>& input, int batch) {
    int totalFloats = input.size();               
    int n = totalFloats / (2 * batch);            
    int totalComplex = totalFloats / 2;           

    hipfftHandle plan;
    // Создаём план для 1D FFT для batch сигналов
    hipfftPlan1d(&plan, n, HIPFFT_C2C, batch);

    // Выделяем память на устройстве для комплексных данных
    hipfftComplex* d_data = nullptr;
    hipMalloc((void**)&d_data, sizeof(hipfftComplex) * totalComplex);

    // Переносим данные на устройство. 
    hipMemcpy(d_data, input.data(), totalFloats * sizeof(float), hipMemcpyHostToDevice);

    // Выполняем прямое преобразование (HIPFFT_FORWARD)
    hipfftExecC2C(plan, d_data, d_data, HIPFFT_FORWARD)

    // Выполняем обратное преобразование (HIPFFT_BACKWARD)
    hipfftExecC2C(plan, d_data, d_data, HIPFFT_BACKWARD);
    
    int threadsPerBlock = 256;
    int blocksPerGrid = (totalComplex + threadsPerBlock - 1) / threadsPerBlock;
    normalizeKernel<<<blocksPerGrid, threadsPerBlock>>>(d_data, totalComplex, n);

    std::vector<float> result(totalFloats);
    hipMemcpy(result.data(), d_data, totalFloats * sizeof(float), hipMemcpyDeviceToHost);

    // Освобождаем ресурсы
    hipFree(d_data);
    hipfftDestroy(plan);

    return result;
}
