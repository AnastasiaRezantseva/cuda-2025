#include "hip/hip_runtime.h"
#include "fft_cufft.h"

__global__ void norm_kernel(float* data, int size, float norm) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < size) {
    data[i] *= norm;
  }
}

std::vector<float> FffCUFFT(const std::vector<float>& input, int batch) {
  int size = input.size();
  std::vector<float> output(size);

  int n = (size >> 1) / batch;
  int cufft_complex_sizeof = sizeof(hipfftComplex) * n * batch;

  hipfftHandle plan;
  hipfftPlan1d(&plan, n, HIPFFT_C2C, batch);

  hipfftComplex* data;
  hipMalloc((void**)&data, cufft_complex_sizeof);

  hipMemcpy(data, input.data(), cufft_complex_sizeof, hipMemcpyHostToDevice);

  hipfftExecC2C(plan, data, data, HIPFFT_FORWARD);
  hipfftExecC2C(plan, data, data, HIPFFT_BACKWARD);

  float norm = 1.0f / (float)n;
  int block_size = 256;
  int num_blocks = (size + block_size - 1) / block_size;
  norm_kernel<<<num_blocks, block_size>>>((float*)data, size, norm);

  hipMemcpy(output.data(), data, cufft_complex_sizeof, hipMemcpyDeviceToHost);

  hipfftDestroy(plan);
  hipFree(data);

  return output;
}