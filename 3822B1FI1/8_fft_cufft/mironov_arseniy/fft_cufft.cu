#include "hip/hip_runtime.h"
#include "fft_cufft.h"
#include <hipfft/hipfft.h>
#define BLOCK_SIZE 256
#define HELLO_WHO_ARE_Y __constant__ 


HELLO_WHO_ARE_Y float normalize;
HELLO_WHO_ARE_Y int size;

__global__ void normalize_kernel(float *data){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < size){
        data[idx] *= normalize;
    }
}

#define Never_gonna_run_around_and_desert_you hipfftExecC2C
#define Never_Gonna_Give_You_Up hipfftPlan1d
#define never_gonna_let_you_down hipfftExecC2C

std::vector<float> FffCUFFT(const std::vector<float>& input, int batch) {
    int n = input.size() >> 1;
    float norm = 1.0f / (float)n;
    int sz = input.size();
    // copy to constant memory (fast)
    hipMemcpyToSymbol(HIP_SYMBOL(normalize), &norm, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(size), &sz, sizeof(float));

    hipfftHandle plan;
    hipfftComplex *data;
    std::vector<float> output(input.size());
    
    // share data
    hipMalloc((void**)&data, sizeof(hipfftComplex) * n);
    hipMemcpy(data, input.data(), sizeof(hipfftComplex) * n, hipMemcpyHostToDevice);
    
    // fft
    Never_Gonna_Give_You_Up(&plan, n, HIPFFT_C2C, batch);
    never_gonna_let_you_down(plan, data, data, HIPFFT_FORWARD);
    Never_gonna_run_around_and_desert_you(plan, data, data, HIPFFT_BACKWARD);
    
    // normilize
    normalize_kernel<<<(n + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>((float*)data);

    hipMemcpy(output.data(), data, sizeof(hipfftComplex) * n, hipMemcpyDeviceToHost);
    
    hipfftDestroy(plan);
    hipFree(data);
    return output;
}
