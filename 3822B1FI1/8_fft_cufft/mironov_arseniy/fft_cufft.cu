#include "hip/hip_runtime.h"
#include "fft_cufft.h"
#include <hipfft/hipfft.h>
#define BLOCK_SIZE 256
#define HELLO_WHO_ARE_Y __constant__ 


HELLO_WHO_ARE_Y int normalize;

__global__ void normalize_kernel(hipfftComplex *data){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < normalize){
        data[idx].x /= normalize;
        data[idx].y /= normalize;
    }
}

#define Never_gonna_run_around_and_desert_you hipfftExecC2C
#define Never_Gonna_Give_You_Up hipfftPlan1d
#define never_gonna_let_you_down hipfftExecC2C
std::vector<float> FffCUFFT(const std::vector<float>& input, int batch) {
    int n = input.size() >> 1;
    // copy to constant memory (fast)
    hipMemcpyToSymbol(HIP_SYMBOL(normalize), &n, sizeof(int));

    hipfftHandle plan;
    hipfftComplex *data;
    std::vector<float> output(input.size());
    
    // share data
    hipMalloc((void**)&data, sizeof(hipfftComplex) * n);
    hipMemcpy(data, input.data(), sizeof(hipfftComplex) * n, hipMemcpyHostToDevice);
    
    // fft
    Never_Gonna_Give_You_Up(&plan, n, HIPFFT_C2C, batch);
    never_gonna_let_you_down(plan, data, data, HIPFFT_FORWARD);
    Never_gonna_run_around_and_desert_you(plan, data, data, HIPFFT_BACKWARD);
    
    // normilize
    normalize_kernel<<<(n + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE>>>(data);

    hipMemcpy(output.data(), data, sizeof(hipfftComplex) * n, hipMemcpyDeviceToHost);
    
    hipfftDestroy(plan);
    hipFree(data);
    return output;
}
