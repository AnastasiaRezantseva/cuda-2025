#include "hip/hip_runtime.h"
#include "fft_cufft.h"
#include <hipfft/hipfft.h>
#define BLOCK_SIZE 256
#define HELLO_WHO_ARE_Y __constant__ 


HELLO_WHO_ARE_Y float normalize;
HELLO_WHO_ARE_Y int size;

__global__ void normalize_kernel(float *data){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx < size){
        data[idx] *= normalize;
    }
}

#define Never_gonna_run_around_and_desert_you hipfftExecC2C
#define Never_Gonna_Give_You_Up hipfftPlan1d
#define never_gonna_let_you_down hipfftExecC2C
std::vector<float> FffCUFFT(const std::vector<float>& input, int batch) {

    
    int sz = input.size();
	int n = sz / (2 * batch);
    float norm = 1.0f / static_cast<float>(n);
    hipMemcpyToSymbol(HIP_SYMBOL(normalize), &norm, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(size), &sz, sizeof(int));

    
    hipfftHandle plan;
	hipfftComplex* data;
    std::vector<float> output(sz);

	hipMalloc(&data,  sz * sizeof(float));
	hipMemcpy(data, input.data(),  sz * sizeof(float), hipMemcpyHostToDevice);

	Never_Gonna_Give_You_Up(&plan, n, HIPFFT_C2C, batch);
    never_gonna_let_you_down(plan, data, data, HIPFFT_FORWARD);
    Never_gonna_run_around_and_desert_you(plan, data, data, HIPFFT_BACKWARD);

	normalize_kernel << <(sz + BLOCK_SIZE - 1) / BLOCK_SIZE, BLOCK_SIZE >> > ((float*)(data));

	hipMemcpy(output.data(), data,  sz * sizeof(float), hipMemcpyDeviceToHost);

	hipfftDestroy(plan);
	hipFree(data);

	return output;
}
