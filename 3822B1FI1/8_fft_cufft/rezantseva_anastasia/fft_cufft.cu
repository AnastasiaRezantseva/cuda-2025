#include "hip/hip_runtime.h"
#include "fft_cufft.h"
#include <cstring>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

#define BLOCK_SIZE 256

__global__ void normalizeKernel(hipfftComplex* data, int totalComplex, float normFactor) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < totalComplex) {
        data[idx].x *= normFactor;
        data[idx].y *= normFactor;
    }
}

std::vector<float> FffCUFFT(const std::vector<float>& input, int batch) {
    int totalFloats = input.size();
    int n = totalFloats / (2 * batch);
    int totalComplex = totalFloats / 2;

    hipfftHandle plan;
    hipfftPlan1d(&plan, n, HIPFFT_C2C, batch);

    hipStream_t stream;
    hipStreamCreate(&stream);
    hipfftSetStream(plan, stream);

    size_t memory = totalFloats * sizeof(float);
    float* h_input_pinned, * h_output_pinned;
    hipHostAlloc(&h_input_pinned, memory, hipHostMallocDefault);
    hipHostAlloc(&h_output_pinned, memory, hipHostMallocDefault);
    std::memcpy(h_input_pinned, input.data(), memory);

    hipfftComplex* d_data = nullptr;
    hipMalloc((void**)&d_data, sizeof(hipfftComplex) * totalComplex);
    hipMemcpyAsync(d_data, h_input_pinned, memory, hipMemcpyHostToDevice, stream);

    hipfftExecC2C(plan, d_data, d_data, HIPFFT_FORWARD);
    hipfftExecC2C(plan, d_data, d_data, HIPFFT_BACKWARD);

    int threadsPerBlock = BLOCK_SIZE;
    int blocksPerGrid = (totalComplex + threadsPerBlock - 1) / threadsPerBlock;
    float normFactor = 1.0f / n;
    normalizeKernel << <blocksPerGrid, threadsPerBlock, 0, stream >> > (d_data, totalComplex, normFactor);

    hipMemcpyAsync(h_output_pinned, d_data, memory, hipMemcpyDeviceToHost, stream);
    hipStreamSynchronize(stream);

    std::vector<float> result(totalFloats);
    std::memcpy(result.data(), h_output_pinned, memory);

    hipFree(d_data);
    hipHostFree(h_input_pinned);
    hipHostFree(h_output_pinned);
    hipStreamDestroy(stream);
    hipfftDestroy(plan);

    return result;
}