#include "hip/hip_runtime.h"
#include "fft_cufft.h"
#include <cstring>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

#define BLOCK_SIZE 256

__global__ void normalizeKernel(float* data, int total_size, float normFactor) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < total_size) {
        data[idx] *= normFactor;
    }
}

std::vector<float> FffCUFFT(const std::vector<float>& input, int batch) {
    int totalFloats = input.size();
    int n = totalFloats / (2 * batch);
    int totalComplex = totalFloats / 2;

    hipfftHandle plan;
    hipfftPlan1d(&plan, n, HIPFFT_C2C, batch);

    size_t memory = totalFloats * sizeof(float);
    hipfftComplex* d_data = nullptr;
    hipMalloc((void**)&d_data, sizeof(hipfftComplex) * totalComplex);
    hipMemcpy(d_data, input.data(), memory, hipMemcpyHostToDevice);

    hipfftExecC2C(plan, d_data, d_data, HIPFFT_FORWARD);
    hipfftExecC2C(plan, d_data, d_data, HIPFFT_BACKWARD);

    int threadsPerBlock = BLOCK_SIZE;
    int blocksPerGrid = (totalComplex + threadsPerBlock - 1) / threadsPerBlock;
    float normFactor = 1.0f / n;
    normalizeKernel << <blocksPerGrid, threadsPerBlock >> > (reinterpret_cast<float*>(d_data), totalFloats, normFactor);

    std::vector<float> result(totalFloats);
    hipMemcpy(result.data(), d_data, memory, hipMemcpyDeviceToHost);

    hipFree(d_data);
    hipfftDestroy(plan);

    return result;
}