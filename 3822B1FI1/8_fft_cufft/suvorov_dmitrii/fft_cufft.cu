#include "hip/hip_runtime.h"
#include "fft_cufft.h"
#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>

__global__ void normalize_kernel(float* buffer, int length, float factor) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < length) {
        buffer[idx] *= factor;
    }
}

std::vector<float> FffCUFFT(const std::vector<float>& srcData, int numBatches) {
    const int dataSize = srcData.size();
    const int fftLen = (dataSize / numBatches) >> 1;
    const int bufferBytes = sizeof(hipfftComplex) * fftLen * numBatches;

    hipfftComplex* gpuBuffer;
    hipfftHandle plan;
    hipfftPlan1d(&plan, fftLen, HIPFFT_C2C, numBatches);
    hipMalloc(&gpuBuffer, bufferBytes);
    hipMemcpy(gpuBuffer, srcData.data(), bufferBytes, hipMemcpyHostToDevice);

    hipfftExecC2C(plan, gpuBuffer, gpuBuffer, HIPFFT_FORWARD);
    hipfftExecC2C(plan, gpuBuffer, gpuBuffer, HIPFFT_BACKWARD);

    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);
    const int threads = prop.maxThreadsPerBlock;
    const int blocks = (dataSize + threads - 1) / threads;
    const float scale = 1.0f / static_cast<float>(fftLen);

    normalize_kernel<<<blocks, threads>>>(
        reinterpret_cast<float*>(gpuBuffer), dataSize, scale);

    std::vector<float> output(dataSize);
    hipMemcpy(output.data(), gpuBuffer, bufferBytes, hipMemcpyDeviceToHost);

    hipfftDestroy(plan);
    hipFree(gpuBuffer);

    return output;
}
